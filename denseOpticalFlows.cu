#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include "opencv2/core.hpp"
#include <opencv2/core/utility.hpp>
#include "opencv2/highgui.hpp"
#include "opencv2/cudaoptflow.hpp"
#include "opencv2/cudaarithm.hpp"

using namespace std;
using namespace cv;
using namespace cv::cuda;

// --- Relative lengths of color transitions, chosen according to perceptual similarity (e.g. one can distinguish 
//     more shades between red and yellow than between yellow and green)
const int RY		= 15;
const int YG		= 6;
const int GC		= 4;
const int CB		= 11;
const int BM		= 13;
const int MR		= 6;
const int NSHADES	= RY + YG + GC + CB + BM + MR;

// --- Color wheel
static Vec3i colorWheel[NSHADES];

/********************************/
/* COMPUTE COLOR WHEEL FUNCTION */
/********************************/
void computeColorWheel() {

	int k = 0;
	for (int i = 0; i < RY; ++i, ++k) colorWheel[k] = Vec3i(255,				255 * i / RY,		0);
	for (int i = 0; i < YG; ++i, ++k) colorWheel[k] = Vec3i(255 - 255 * i / YG, 255,				0);
	for (int i = 0; i < GC; ++i, ++k) colorWheel[k] = Vec3i(0,					255,				255 * i / GC);
	for (int i = 0; i < CB; ++i, ++k) colorWheel[k] = Vec3i(0,					255 - 255 * i / CB, 255);
	for (int i = 0; i < BM; ++i, ++k) colorWheel[k] = Vec3i(255 * i / BM,		0,					255);
	for (int i = 0; i < MR; ++i, ++k) colorWheel[k] = Vec3i(255,				0,					255 - 255 * i / MR);
}

/********************/
/* VALID FLOW CHECK */
/********************/
inline bool isFlowValid(Point2f u) { return !cvIsNaN(u.x) && !cvIsNaN(u.y) && fabs(u.x) < 1e9 && fabs(u.y) < 1e9; }

/*****************************/
/* RETURN HUE COLOR FUNCTION */
/*****************************/
static Vec3b returnCOLOR(float ux, float uy) {

	// --- Displacement: distance from origin
	const float rad				= sqrt(ux * ux + uy * uy);
	// --- Displacement: angular position normalized by pi (angular position comprised in (-1, 1)
	const float angleNormalized	= atan2(-uy, -ux) / (float)CV_PI;

	// --- Divides the full circle into NSHADES slices and computes the position of the displacement vector
	//     within such slices
	const float shadingPosition	= (angleNormalized + 1.0f) / 2.0f * (NSHADES - 1);
	// --- Beginning slice index
	const int	shade0	= static_cast<int>(shadingPosition);
	// --- Ending slice index
	const int	shade1	= (shade0 + 1) % NSHADES;
	// --- Offset wrt beginning slice index
	const float f	= shadingPosition - shade0;

	Vec3b displacementColor;

	for (int RGBcol = 0; RGBcol < 3; RGBcol++)
	{
		// --- Computes normalized RGB color corresponding to initial shade
		const float col0 = colorWheel[shade0][RGBcol] / 255.0f;
		// --- Computes normalized RGB color corresponding to final shade
		const float col1 = colorWheel[shade1][RGBcol] / 255.0f;

		// --- Computes hue of HSV according to angular position of displacement
		float col = (1 - f) * col0 + f * col1;

		if (rad <= 1)
			// --- Change saturation with radius. For rad = 0, col = 1; for rad = 1, col = col.
			col = 1 - rad * (1 - col); 
		else
			// --- Radius out of range
			col *= .75; 

		displacementColor[RGBcol] = static_cast<uchar>(255.0 * col);
	}

	return displacementColor;
}

/**************************/
/* COLORING FLOW FUNCTION */
/**************************/
static void colorOpticalFlow(const Mat_<float> &h_dx, const Mat_<float> &h_dy, Mat &h_coloredFlow, float maxmotion)
{
	// --- Creates a colored flow of the same size of the flow and sets it to all zeros
	h_coloredFlow.create(h_dx.size(), CV_8UC3);
	h_coloredFlow.setTo(Scalar::all(0));

	for (int y = 0; y < h_dx.rows; ++y) {
		for (int x = 0; x < h_dx.cols; ++x) {
			
			Point2f u(h_dx(y, x), h_dy(y, x));

			// --- In the flow is valid, returns the color associated to the displacement
			if (isFlowValid(u)) h_coloredFlow.at<Vec3b>(y, x) = returnCOLOR(u.x / maxmotion, u.y / maxmotion); }}}

/**********************************/
/* COMPUTE FLOW AND SHOW FUNCTION */
/**********************************/
static void computeFlowAndShow(const char *name, const GpuMat &d_opticalFlow)
{
	// --- Split the x and y components of the displacement of a two channel matrix into an array of matrices
	GpuMat planes[2];
	cuda::split(d_opticalFlow, planes);

	// --- Copy the x and y flow components to CPU
	Mat opticalFlowx(planes[0]);
	Mat opticalFlowy(planes[1]);

	Mat out;
	colorOpticalFlow(opticalFlowx, opticalFlowy, out, 10);

	imshow(name, out);
}

/**********************/
/* FILECHECK FUNCTION */
/**********************/
int fileCheck(Mat &im0, Mat &im1, string &filename1, string &filename2) {
	
	if (im0.empty())
	{
		cerr << "Image file [" << filename1 << "] can't be opened. Please, check." << endl;
		return -1;
	}
	
	if (im1.empty())
	{
		cerr << "Image file [" << filename2 << "] can't be opened. Please, check." << endl;
		return -1;
	}

	if (im1.size() != im0.size())
	{
		cerr << "Images are not of equal size. Please, check." << endl;
		return -1;
	}
}

/********/
/* MAIN */
/********/
int main() {
	
	// --- Images file names
	// --- https://ccv.wordpress.fos.auckland.ac.nz/data/stereo-pairs/
	string filename1 = "./rect_0384_c1.tif";
	string filename2 = "./rect_0385_c1.tif";

	// --- Loading images into OpenCV matrices
	Mat im0 = imread(filename1, IMREAD_GRAYSCALE);
	Mat im1 = imread(filename2, IMREAD_GRAYSCALE);

	// --- Checking wether files can be opened or images have the same size
	const int fileCheckInt = fileCheck(im0, im1, filename1, filename2);
	if (fileCheckInt == -1) return -1;
		 
	// --- Moving images from CPU to GPU
	GpuMat d_im0(im0);
	GpuMat d_im1(im1);

	GpuMat d_opticalFlow(im0.size(), CV_32FC2);

	// --- Compute color wheel
	computeColorWheel();

	/*************/
	/* FARNEBACK */
	/*************/
	// ---  cuda::FarnebackOpticalFlow::create(int numLevels=5, double pyrScale=0.5, bool fastPyramids=false, int winSize=13, int numIters=10, int polyN=5, double polySigma=1.1, int flags=0)
	//		numLevels			= number of pyramid layers including the initial image; levels=1 means that no extra 
	//							  layers are created and only the original images are used.
	//		pyr_scale			= parameter, specifying the image scale (<1) to build pyramids for each image; 
	//							  pyr_scale=0.5 means a classical pyramid, where each next layer is twice smaller 
	//							  than the previous one.
	//		fastPyramids		= use fast pyramids approach
	//		winsize				= averaging window size; larger values increase the algorithm robustness to image 
	//							  noise and give more chances for fast motion detection, but yield more blurred 
	//							  motion field.
	//      numIters			= number of iterations the algorithm does at each pyramid level.
	//		polyN 				= size of the pixel neighborhood used to find polynomial expansion in each pixel; 
	//							  larger values mean that the image will be approximated with smoother surfaces, 
	//						      yielding more robust algorithm and more blurred motion field, typically poly_n=5 or 7.
	//      polySigma			= standard deviation of the Gaussian that is used to smooth derivatives used as a 
	//							  basis for the polynomial expansion; for polyN=5, you can set polySigma=1.1, 
	//							  for polyN=7, a good value would be polySigma=1.5.
	//		flags				= operation flags that can be a combination of the following:
	//							  OPTFLOW_USE_INITIAL_FLOW uses the input flow as an initial flow approximation.
	//							  OPTFLOW_FARNEBACK_GAUSSIAN uses the Gaussian winsize x winsize filter instead of 
	//							  a box filter of the same size for optical flow estimation; usually, this option 
	//							  gives z more accurate flow than with a box filter, at the cost of lower speed; normally, winsize for a Gaussian window should be set to a larger value to achieve the same level of robustness.
	Ptr<cuda::FarnebackOpticalFlow>		farn = cuda::FarnebackOpticalFlow::create(6, 0.4, false, 13, 15, 5, 1.1, 256);
	{
		farn->calc(d_im0, d_im1, d_opticalFlow);

		computeFlowAndShow("Farnebäck", d_opticalFlow);
	}

	/*********************/
	/* BROX OPTICAL FLOW */
	/*********************/
	// ---	cuda::BroxOpticalFlow::create(double alpha=0.197, double gamma=50.0, double scale_factor=0.8, int inner_iterations=5, int outer_iterations=150, int solver_iterations=10)
	//		alpha				= flow smoothness functional weight
	//		gamma				= gradient constancy importance functional weight
	//		scale_factor		= pyramid scale factor belonging to (0,1)
	//		inner_iterations	= number of lagged non-linearity iterations (inner loop)
	//		outer_iterations	= number of pyramid levels
	//		solver_iterations	= number of linear system solver iterations
	Ptr<cuda::BroxOpticalFlow>			brox = cuda::BroxOpticalFlow::create(0.197f, 50.0f, 0.8f, 10, 77, 10);
	{
		GpuMat d_im0Scaled;
		GpuMat d_im1Scaled;

		d_im0.convertTo(d_im0Scaled, CV_32F, 1.0 / 255.0);
		d_im1.convertTo(d_im1Scaled, CV_32F, 1.0 / 255.0);

		brox->calc(d_im0Scaled, d_im1Scaled, d_opticalFlow);

		computeFlowAndShow("Brox et al.", d_opticalFlow);
	}

	/**************/
	/* DUAL TV-L1 */
	/**************/
	// ---  cuda::OpticalFlowDual_TVL1::create(double tau=0.25, double lambda=0.15, double theta=0.3, int nscales=5, int warps=5, double epsilon=0.01, int iterations=300, double scaleStep=0.8, double gamma=0.0, bool useInitialFlow=false)
	//		tau					= time step of the numerical scheme
	//		lambda				= weight parameter for the data term, attachment parameter. This is the most relevant 
	//                            parameter, which determines the smoothness of the output. The smaller this 
	//							  parameter is, the smoother the solutions we obtain. It depends on the range of 
	//							  motions of the images, so its value should be adapted to each image sequence.
	//		theta				= weight parameter for (u - v)^2, tightness parameter. It serves as a link between 
	//							  the attachment and the regularization terms. In theory, it should have a small 
	//							  value in order to maintain both parts in correspondence. The method is stable for 
	//							  a large range of values of this parameter.
	//		nscales				= number of scales used to create the pyramid of images.
	//		warps				= number of warpings per scale. Represents the number of times that I1(x+u0) and 
	//						      grad( I1(x+u0) ) are computed per scale. This is a parameter that assures the 
	//							  stability of the method. It also affects the running time, so it is a compromise 
	//							  between speed and accuracy.
	//		epsilon				= stopping criterion threshold used in the numerical scheme, which is a trade-off 
	//							  between precision and running time. A small value will yield more accurate 
	//							  solutions at the expense of a slower convergence.	
	//		iterations			= stopping criterion iterations number used in the numerical scheme.
	//		scaleStep			= Step between scales (<1).
	//		gamma				= *q +.
	//		useInitialFlow		= Use initial flow.
	Ptr<cuda::OpticalFlowDual_TVL1>		tvl1 = cuda::OpticalFlowDual_TVL1::create();
	// Ptr<cuda::OpticalFlowDual_TVL1>		tvl1 = cuda::OpticalFlowDual_TVL1::create(0.25, 0.15, 0.3, 5, 5, 0.01, 300, 0.8, 10.0, false);
	{
		tvl1->calc(d_im0, d_im1, d_opticalFlow);

		computeFlowAndShow("TVL1", d_opticalFlow);
	}

	/****************/
	/* LUCAS-KANADE */
	/****************/
	// ---  cv::cuda::DensePyrLKOpticalFlow::create(Size winSize = Size(13, 13), int maxLevel = 3, int iters = 30, bool useInitialFlow = false)
	//		winSize				= size of the search window at each pyramid level
	//		maxLevel			= 0-based maximal pyramid level number; if set to 0, pyramids are not used (single 
	//							  level), if set to 1, two levels are used, and so on; if pyramids are passed to 
	//							  input then algorithm will use as many levels as pyramids have but no more than 
	//							  maxLevel.
	//		iters				= number of iterations according to http://www.ieee-hpec.org/2014/CD/index_htm_files/FinalPapers/98.pdf
	//		useInitialFlow		= exploit User's provided initial flow as starting guess
	Ptr<cuda::DensePyrLKOpticalFlow>	lk = cuda::DensePyrLKOpticalFlow::create(Size(7, 7));
	{
		lk->calc(d_im0, d_im1, d_opticalFlow);

		computeFlowAndShow("Lucas-Kanade", d_opticalFlow);
	}

	imshow("Frame 0", im0);
	imshow("Frame 1", im1);
	waitKey();

	return 0;
}

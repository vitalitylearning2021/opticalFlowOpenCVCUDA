#include "hip/hip_runtime.h"
#include <  iostream>    
#include "opencv2\objdetect\objdetect.hpp"
#include "opencv2\highgui\highgui.hpp"
#include "opencv2\imgproc\imgproc.hpp"
#include "opencv2\cudaobjdetect.hpp"
#include "opencv2\cudaimgproc.hpp"
#include "opencv2\cudawarping.hpp"
#include <  opencv2\bgsegm.hpp>  
#include <  opencv2\cudabgsegm.hpp>  
#include <  opencv2\cudaoptflow.hpp>  
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"

using namespace cv;
using namespace std;
using namespace cv::cuda;

/*********************************/
/* DEVICE TO HOST COPY FUNCTIONS */
/*********************************/
static void device2Host(const GpuMat &d_A, vector<Point2f> &h_A) {
	h_A.resize(d_A.cols);
	Mat mat(1, d_A.cols, CV_32FC2, (void *)&h_A[0]);
	d_A.download(mat); }

static void device2Host(const GpuMat &d_A, vector<uchar> &h_A) {
	h_A.resize(d_A.cols);
	Mat mat(1, d_A.cols, CV_8UC1, (void *)&h_A[0]);
	d_A.download(mat); }

/******************************/
/* DRAW OPTICAL FLOW FUNCTION */
/******************************/
static void drawFlow(Mat &frame, const vector<Point2f> &previousPoints, const vector<Point2f> &nextPoints, const vector<uchar> &status, Scalar line_color = Scalar(0, 0, 255)) {

	// --- Loop over all the points
	for (size_t i = 0; i < previousPoints.size(); ++i) {
		
		// --- Check if point status is ok
		if (status[i]) {
			
			// --- Set line thickness
			int line_thickness = 1;

			// --- Set previous and next points
			Point p = previousPoints[i];
			Point q = nextPoints[i];

			// --- Find angle and length of arrow
			double angle = atan2((double)p.y - q.y, (double)p.x - q.x);
			double hypotenuse = sqrt((double)(p.y - q.y)*(p.y - q.y) + (double)(p.x - q.x)*(p.x - q.x));

			if (hypotenuse < 1.0) continue;

			// --- If the length of the arrow is less than 1, then lengthen the arrow by a factor of three.
			q.x = (int)(p.x - 3 * hypotenuse * cos(angle));
			q.y = (int)(p.y - 3 * hypotenuse * sin(angle));

			// --- Define arrow line
			line(frame, p, q, line_color, line_thickness);

			// --- Draw the tips of the arrow. Some scaling is operated so that the tips look proportional to the 
			//     main line of the arrow.
			p.x = (int)(q.x + 9 * cos(angle + CV_PI / 4));
			p.y = (int)(q.y + 9 * sin(angle + CV_PI / 4));
			line(frame, p, q, line_color, line_thickness);

			p.x = (int)(q.x + 9 * cos(angle - CV_PI / 4));
			p.y = (int)(q.y + 9 * sin(angle - CV_PI / 4));
			line(frame, p, q, line_color, line_thickness);
		}
	}
}

/********/
/* MAIN */
/********/
void main() {
	
	// --- CPU images
	Mat im0, im1;

	// --- GPU images
	cuda::GpuMat d_im0Gray, d_im1Gray, d_frame0, d_frame1;

	// --- https://pixabay.com/videos/car-traffic-daytime-driving-on-road-16849/
	VideoCapture cap("Car - 16849.mp4");
	cap >> im0;
	if (im0.empty()) return;

	// --- Scaling factor
	//double scale = 800. / im0.cols;
	double scale = 1;

	// --- First image
	d_frame0.upload(im0);
	cuda::resize(d_frame0, d_im0Gray, Size(d_frame0.cols * scale, d_frame0.rows * scale));
	cuda::cvtColor(d_im0Gray, d_im0Gray, COLOR_BGR2GRAY);

	/***********************************/
	/* SPOTTING GOOD FEATURES TO TRACK */
	/***********************************/
	int		maxCorners		= 4000;
	double	qualityLevel	= 0.01;
	double	minDistance		= 0;
	int		blockSize		= 5;
	Ptr<cuda::CornersDetector> detector = cuda::createGoodFeaturesToTrackDetector(d_im0Gray.type(), maxCorners, qualityLevel, minDistance, blockSize);

	/**************************************/
	/* LUCAS-KANADE'S SPARSE OPTICAL FLOW */
	/**************************************/
	cuda::GpuMat d_previousPoints;
	cuda::GpuMat d_nextPoints;
	cuda::GpuMat d_status;

	int		winSize			= 21;
	int		maxLevel		= 3;
	int		iters			= 30;
	Ptr<cuda::SparsePyrLKOpticalFlow> d_pyrLK = cuda::SparsePyrLKOpticalFlow::create(Size(winSize, winSize), maxLevel, iters);

	while (1) {

		// --- Get new image
		cap >> im1;
		if (im1.empty()) break;

		d_frame1.upload(im1);
		cuda::resize(d_frame1, d_im1Gray, Size(d_frame1.cols * scale, d_frame1.rows * scale));
		d_im1Gray.download(im1);
		// --- Converts image to gray scale
		cuda::cvtColor(d_im1Gray, d_im1Gray, COLOR_BGR2GRAY);

		// --- Good features to track: detection
		detector->detect(d_im0Gray, d_previousPoints);
		
		// --- Compute optical flow
		d_pyrLK->calc(d_im0Gray, d_im1Gray, d_previousPoints, d_nextPoints, d_status);

		// --- Swap old and new grey-scale images
		d_im0Gray = d_im1Gray;

		// --- Copies from device to host
		vector<Point2f> h_previousPoints(d_previousPoints.cols);
		device2Host(d_previousPoints, h_previousPoints);
		
		vector<Point2f> h_nextPoints(d_nextPoints.cols);
		device2Host(d_nextPoints, h_nextPoints);
		
		vector<uchar> status(d_status.cols);
		device2Host(d_status, status);
		
		// --- Draw optical flow
		drawFlow(im1, h_previousPoints, h_nextPoints, status, Scalar(255, 0, 0));
		imshow("PyrLK [Sparse]", im1);

		if (waitKey(10) > 0) break;
	}

}


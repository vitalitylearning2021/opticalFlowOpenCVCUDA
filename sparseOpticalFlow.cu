#include "hip/hip_runtime.h"
/***********************/
/* LUCAS-KANADE SPARSE */
/***********************/

#include <iostream>
#include <vector>

#include <opencv2/core.hpp>
#include <opencv2/core/utility.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/video.hpp>
#include <opencv2/cudaoptflow.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>

using namespace std;
using namespace cv;
using namespace cv::cuda;

/*********************************/
/* DEVICE TO HOST COPY FUNCTIONS */
/*********************************/
static void device2Host(const GpuMat &d_A, vector<Point2f> &h_A) {
	h_A.resize(d_A.cols);
	Mat mat(1, d_A.cols, CV_32FC2, (void *)&h_A[0]);
	d_A.download(mat); }

static void device2Host(const GpuMat &d_A, vector<uchar> &h_A) {
	h_A.resize(d_A.cols);
	Mat mat(1, d_A.cols, CV_8UC1, (void *)&h_A[0]);
	d_A.download(mat); }

/******************************/
/* DRAW OPTICAL FLOW FUNCTION */
/******************************/
static void drawFlow(Mat &frame, const vector<Point2f> &previousPoints, const vector<Point2f> &nextPoints, const vector<uchar> &status, Scalar line_color = Scalar(0, 0, 255)) {

	// --- Loop over all the points
	for (size_t i = 0; i < previousPoints.size(); ++i) {
		
		// --- Check if point status is ok
		if (status[i]) {
			
			// --- Set line thickness
			int line_thickness = 1;

			// --- Set previous and next points
			Point p = previousPoints[i];
			Point q = nextPoints[i];

			// --- Find angle and length of arrow
			double angle = atan2((double)p.y - q.y, (double)p.x - q.x);
			double hypotenuse = sqrt((double)(p.y - q.y)*(p.y - q.y) + (double)(p.x - q.x)*(p.x - q.x));

			if (hypotenuse < 1.0) continue;

			// --- If the length of the arrow is less than 1, then lengthen the arrow by a factor of three.
			q.x = (int)(p.x - 3 * hypotenuse * cos(angle));
			q.y = (int)(p.y - 3 * hypotenuse * sin(angle));

			// --- Define arrow line
			line(frame, p, q, line_color, line_thickness);

			// --- Draw the tips of the arrow. Some scaling is operated so that the tips look proportional to the 
			//     main line of the arrow.
			p.x = (int)(q.x + 9 * cos(angle + CV_PI / 4));
			p.y = (int)(q.y + 9 * sin(angle + CV_PI / 4));
			line(frame, p, q, line_color, line_thickness);

			p.x = (int)(q.x + 9 * cos(angle - CV_PI / 4));
			p.y = (int)(q.y + 9 * sin(angle - CV_PI / 4));
			line(frame, p, q, line_color, line_thickness);
		}
	}
}

/********/
/* MAIN */
/********/
int main() {

	// --- Images file names
	// --- https://ccv.wordpress.fos.auckland.ac.nz/data/stereo-pairs/
	string filename0 = "./rect_0384_c1.tif";
	string filename1 = "./rect_0385_c1.tif";

	Mat im0 = imread(filename0);
	Mat im1 = imread(filename1);

	cout << "Image size: " << im0.cols << " x " << im0.rows << endl;

	Mat im0Gray;
	//Mat im1Gray;

	// --- Converts images im0 and im1 to gray scale
	cv::cvtColor(im0, im0Gray, COLOR_BGR2GRAY);
	//cv::cvtColor(im1, im1Gray, COLOR_BGR2GRAY);

	/***********************************/
	/* SPOTTING GOOD FEATURES TO TRACK */
	/***********************************/
	GpuMat d_im0Gray(im0Gray);
	GpuMat d_previousPoints;

	// --- https://opencv-python-tutroals.readthedocs.io/en/latest/py_tutorials/py_feature2d/py_features_meaning/py_features_meaning.html
	// --- Ptr<CornersDetector> cv::cuda::createGoodFeaturesToTrackDetector(int srcType, int maxCorners = 1000,
	//		   					double qualityLevel = 0.01, double minDistance = 0.0, int blockSize = 3,
	//							bool useHarrisDetector = false, double harrisK = 0.04)	
	// --- srcType				Input source type. Only CV_8UC1 and CV_32FC1 are supported for now.
	// --- maxCorners			Maximum number of corners to return. If there are more corners than are found, the strongest of them is returned.
	// --- qualityLevel			Parameter characterizing the minimal accepted quality of image corners.The parameter 
	//							value is multiplied by the best corner quality measure, which is the minimal 
	//							eigenvalue (see cornerMinEigenVal) or the Harris function response (see cornerHarris).The corners with the quality measure less than the product are rejected.For example, if the best corner has the quality measure = 1500, and the qualityLevel = 0.01, then all the corners with the quality measure less than 15 are rejected.
	// --- minDistance			Minimum possible Euclidean distance between the returned corners.
	// --- blockSize			Size of an average block for computing a derivative covariation matrix over each pixel neighborhood.See cornerEigenValsAndVecs .
	// --- useHarrisDetector	Parameter indicating whether to use a Harris detector (see cornerHarris) or 
	//							cornerMinEigenVal.
	// --- harrisK				Free parameter of the Harris detector.

	int		maxCorners = 4000;
	double	qualityLevel = 0.01;
	double	minDistance = 0;
	int		blockSize = 5;

	Ptr<cuda::CornersDetector> detector = cuda::createGoodFeaturesToTrackDetector(d_im0Gray.type(), maxCorners, qualityLevel,
		minDistance, blockSize);
	detector->detect(d_im0Gray, d_previousPoints);

	/**************************************/
	/* LUCAS-KANADE'S SPARSE OPTICAL FLOW */
	/**************************************/
	GpuMat d_frame0(im0);
	GpuMat d_frame1(im1);
	//GpuMat d_frame1Gray(im1Gray);
	GpuMat d_nextPoints;
	GpuMat d_status;
	//GpuMat d_flow(im0.size(), CV_32FC2);

	int		winSize = 21;
	int		maxLevel = 3;
	int		iters = 30;
	Ptr<cuda::SparsePyrLKOpticalFlow> d_pyrLK_sparse = cuda::SparsePyrLKOpticalFlow::create(Size(winSize, winSize), maxLevel, iters);

	// --- SparseOpticalFlow::calc(InputArray prevImg, InputArray nextImg, InputArray prevPts, InputOutputArray	
	//		nextPts, OutputArray status, OutputArray err = cv::noArray())
	// --- prevImg				First input image.
	// --- nextImg				Second input image of the same size and the same type as prevImg.
	// --- prevPts				Vector of 2D points for which the flow needs to be found.
	// --- nextPts				Output vector of 2D points containing the calculated new positions of input features 
	//							in the second image.
	// --- status				Output status vector.Each element of the vector is set to 1 if the flow for the 
	//							corresponding features has been found. Otherwise, it is set to 0.
	// --- err					Optional output vector that contains error response for each point (inverse 
	//							confidence).
	d_pyrLK_sparse->calc(d_frame0, d_frame1, d_previousPoints, d_nextPoints, d_status);

	// --- Copies from device to host
	vector<Point2f> h_previousPoints(d_previousPoints.cols);
	device2Host(d_previousPoints, h_previousPoints);

	vector<Point2f> h_nextPoints(d_nextPoints.cols);
	device2Host(d_nextPoints, h_nextPoints);

	vector<uchar> status(d_status.cols);
	device2Host(d_status, status);

	// --- Draw optical flow
	namedWindow("PyrLK [Sparse]", WINDOW_NORMAL);
	drawFlow(im0, h_previousPoints, h_nextPoints, status, Scalar(255, 0, 0));
	imshow("PyrLK [Sparse]", im0);

	waitKey(0);

	return 0;
}
